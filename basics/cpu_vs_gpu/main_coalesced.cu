#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

float *alloc_matrix_1D(const int n, const int m) {
  float *matrix = (float *)malloc(sizeof(float) * n * m);
  return matrix;
}

void print_matrix_1D(const int n, const int m, const float *matrix) {
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < m; ++j) {
      printf("%.2f\t", matrix[i * m + j]);
    }
    printf("\n");
  }
}

__global__ void transpose_matrix_coalesced(const int n, const int m,
                                           const float *origin, float *result) {

  __shared__ float tile[16][16 + 1];
  const int i = blockIdx.y * 16 + threadIdx.y;
  const int j = blockIdx.x * 16 + threadIdx.x;

  if (i < n && j < m) {
    tile[threadIdx.y][threadIdx.x] = origin[j * m + i];
  }

  __syncthreads();

  if (i < n && j < m) {
    result[j * n + i] = tile[threadIdx.x][threadIdx.y];
  }
}

float *run_transpose_coalesced(const int n, const int m,
                               const float *host_origin) {
  float *dev_origin, *dev_result, *host_result;
  host_result = alloc_matrix_1D(m, n);
  hipMalloc(&dev_origin, sizeof(float) * n * m);
  hipMalloc(&dev_result, sizeof(float) * m * n);

  hipMemcpy(dev_origin, host_origin, n * m * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 blockSize(16, 16);
  dim3 gridSize((n + blockSize.x - 1) / blockSize.x,
                (m + blockSize.y - 1) / blockSize.y);

  transpose_matrix_coalesced<<<gridSize, blockSize>>>(n, m, dev_origin,
                                                      dev_result);
  hipDeviceSynchronize();
  hipMemcpy(host_result, dev_result, m * n * sizeof(float),
             hipMemcpyDeviceToHost);
  hipFree(dev_origin);
  hipFree(dev_result);
  return host_result;
}

void init_matrix_1D(const int n, const int m, const int base, float *matrix) {
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < m; ++j) {
      matrix[i * m + j] = base * i * j + i + 2 * j;
    }
  }
}

void free_matrix_1D(float *matrix) { free(matrix); }

int main(int argc, char **argv) {
  printf("Hello, world!\n");
  int n = atol(argv[1]);
  int m = atol(argv[2]);
  float *matA = alloc_matrix_1D(n, m);
  float *matB;
  init_matrix_1D(n, m, 2, matA);
  matB = run_transpose_coalesced(n, m, matA);
  free_matrix_1D(matA);
  free_matrix_1D(matB);
  return 0;
}
